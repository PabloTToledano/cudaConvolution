#include "hip/hip_runtime.h"
#include <cv.h>
#include <highgui.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define RED 2
#define GREEN 1
#define BLUE 0

#define MASK_WIDTH 3

__constant__ char M[MASK_WIDTH*MASK_WIDTH];

using namespace cv;

//funcion device
__device__ unsigned char clamp(int value){
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return (unsigned char)value;
}

//funcion global de filtro de sobel
__global__ void sobelFilter(unsigned char *imageInput, int width, int height, \
        unsigned int maskWidth,unsigned char *imageOutput){

    unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;

    int Pvalue = 0;

    int N_start_point_row = row - (maskWidth/2);
    int N_start_point_col = col - (maskWidth/2);

    for(int i = 0; i < maskWidth; i++){
        for(int j = 0; j < maskWidth; j++ ){
            if((N_start_point_col + j >=0 && N_start_point_col + j < width) \
                    &&(N_start_point_row + i >=0 && N_start_point_row + i < height)){
                Pvalue += imageInput[(N_start_point_row + i)*width+(N_start_point_col + j)] * M[i*maskWidth+j];
            }
        }
    }
    imageOutput[row*width+col] = clamp(Pvalue);
}

//funcion global de escala de grises
__global__ void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if((row < height) && (col < width)){
        imageOutput[row*width+col] = imageInput[(row*width+col)*3+RED]*0.299 + imageInput[(row*width+col)*3+GREEN]*0.587 \
                                     + imageInput[(row*width+col)*3+BLUE]*0.114;
    }
}

//MENU
int main(int argc, char **argv){
    hipError_t error = hipSuccess;
    clock_t start, end, startGPU, endGPU;
    double cpu_time_used, gpu_time_used;
    char h_M[] = {-1,0,1,-2,0,2,-1,0,1};
    unsigned char *dataRawImage, *d_dataRawImage, *d_imageOutput, *h_imageOutput, *d_sobelOutput;
  	
    //creamos la matriz de la imagen de entrada
    Mat image;
    //llamamos la imagen
    image = imread("./inputs/img5.jpg", 1);
  	
    //algoritmo secuencial con OpenCV
    start = clock();
    Mat gray_image_opencv, grad_x;
    cvtColor(image, gray_image_opencv, CV_BGR2GRAY);
    Sobel(gray_image_opencv,grad_x,CV_8UC1,1,0,3,1,0,BORDER_DEFAULT);
    end = clock();
    //fin algoritmo secuencial con OpenCV
  
    Size s = image.size();
		
    //inicializamos variables
    int width = s.width;
    int height = s.height;
    int size = sizeof(unsigned char)*width*height*image.channels();
    int sizeGray = sizeof(unsigned char)*width*height;
		
    //Reserva de Memoria para d_dataRawImage
    dataRawImage = (unsigned char*)malloc(size);
    error = hipMalloc((void**)&d_dataRawImage,size);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_dataRawImage\n");
        exit(-1);
    }

    //Reserva de Memoria para d_imageOutput
    h_imageOutput = (unsigned char *)malloc(sizeGray);
    error = hipMalloc((void**)&d_imageOutput,sizeGray);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_imageOutput\n");
        exit(-1);
    }

    //Reserva de Memoria para d_sobelOutput
    error = hipMalloc((void**)&d_sobelOutput,sizeGray);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_sobelOutput\n");
        exit(-1);
    }

    dataRawImage = image.data;
		
    //Algoritmo Paralelo con CUDA
    startGPU = clock();

    error = hipMemcpy(d_dataRawImage,dataRawImage,size, hipMemcpyHostToDevice);
    if(error != hipSuccess){
        printf("Error copiando los datos de dataRawImage a d_dataRawImage \n");
        exit(-1);
    }

    error = hipMemcpyToSymbol(HIP_SYMBOL(M),h_M,sizeof(char)*MASK_WIDTH*MASK_WIDTH);
    if(error != hipSuccess){
        printf("Error copiando los datos de h_M a d_M \n");
        exit(-1);
    }

    int blockSize = 32;
    dim3 dimBlock(blockSize,blockSize,1);
    dim3 dimGrid(ceil(width/float(blockSize)),ceil(height/float(blockSize)),1);
    //llamamos la funcion de escala de grises
    img2gray<<<dimGrid,dimBlock>>>(d_dataRawImage,width,height,d_imageOutput);
    //Sincronizamos
    hipDeviceSynchronize();
    //llamamos la funcion de filtro de sobel
    sobelFilter<<<dimGrid,dimBlock>>>(d_imageOutput,width,height,3,d_sobelOutput);
    hipMemcpy(h_imageOutput,d_sobelOutput,sizeGray,hipMemcpyDeviceToHost);

    endGPU = clock();
    //fin algoritmo Paralelo con CUDA
  	
    //creamos la matriz de la imagen en sobel
    Mat gray_image;
    gray_image.create(height,width,CV_8UC1);
    gray_image.data = h_imageOutput;

  
    //imprimir las imagenes en full color, escala de grises o filtro de sobel(secuencial o paralelo)	
			
    //imwrite("./outputs/1088012385.png",image);		//full color
    //imwrite("./outputs/1088012385.png",gray_image_opencv);	//grises secuencial
    //imwrite("./outputs/1088012385.png",grad_x);		//sobel secuencial
      imwrite("./outputs/1088012385.png",gray_image);		//sobel del paralelo
		
  
    //imprimir tiempos de ejecucion
    gpu_time_used = ((double) (endGPU - startGPU)) / CLOCKS_PER_SEC;
    printf("Tiempo Algoritmo Paralelo CUDA: %.10f\n",gpu_time_used);
    cpu_time_used = ((double) (end - start)) /CLOCKS_PER_SEC;
    printf("Tiempo Algoritmo Secuencial OpenCV: %.10f\n",cpu_time_used);
    printf("La aceleración obtenida es de %.10fX\n",cpu_time_used/gpu_time_used);

    //limpiar memoria
    hipFree(d_dataRawImage);
    hipFree(d_imageOutput);
    hipFree(M);
    hipFree(d_sobelOutput);
    return 0;
}